#include "hip/hip_runtime.h"
#include "game.h"

#include <thrust/device_vector.h>

#include "cuda.cuh"

__global__ void step(utils::matrix_wrapper<std::span<game::tile>> grid, float time)
	{
	const auto coords{utils::cuda::kernel::coordinates::total::vec2()};
	if (!grid.validate_coords(coords)) { return; }

	auto& tile   {grid[coords]};
	auto& terrain{tile.terrain};
	auto& plant  {tile.plant  };

	terrain.step(time);
	plant  .step(terrain, time);
	}

/// <summary>
/// 
/// </summary>
/// <param name="grid"></param>
/// <param name="delta">Delta is [-1, 0, 1]</param>
/// <returns></returns>
__global__ void step_distribution(utils::matrix_wrapper<std::span<game::tile>> grid, utils::math::vec2u delta)
	{
	const auto coords{utils::cuda::kernel::coordinates::total::vec2()};
	if (!grid.validate_coords(coords)) { return; }

	auto& tile{grid[coords]};
	auto& plant{tile.plant};
	auto& neighbour_tile {grid[game::coords::tile_neighbour(coords, grid.sizes(), delta)]};
	auto& neighbour_plant{neighbour_tile.plant};

	plant.step_distribution(neighbour_plant);
	}
__global__ void step_recover_undistributed(utils::matrix_wrapper<std::span<game::tile>> grid)
	{
	const auto coords{utils::cuda::kernel::coordinates::total::vec2()};
	if (!grid.validate_coords(coords)) { return; }

	auto& tile{grid[coords]};
	auto& plant{tile.plant};

	plant.step_recover_undistributed();
	}

__global__ void build_on_tile(utils::matrix_wrapper<std::span<game::tile>> grid, utils::math::vec2s coords, float absorption)
	{
	grid[coords].plant.humidity      = .5f;
	grid[coords].plant.humidity_next = .5f;
	grid[coords].plant.absorption    = absorption;
	}

namespace game
	{
	game::game(const ::game::data_cpu& data_cpu) : data_cpu{data_cpu}, data_gpu{data_cpu} {}

	data_gpu::data_gpu(const data_cpu& data_cpu) :
		grid{data_cpu.grid.begin(), data_cpu.grid.end()},
		grid_kernel_side{[this, &data_cpu]()
			{
			std::span<tile> device_span{thrust::raw_pointer_cast(grid.data()), grid.size()};
			utils::matrix_wrapper<std::span<tile>> grid_kernel_side{data_cpu.grid.sizes(), device_span};
			return grid_kernel_side;
			}()}
		{}
	data_gpu::~data_gpu() = default;

	void game::step(float delta_time) noexcept
		{
		data_cpu.time = data_cpu.next_time;
		data_cpu.next_time += delta_time;

		utils::cuda::device::params_t kernel_params
			{
			.threads{utils::cuda::device::params_t::threads_t::deduce({16u, 16u}, {data_cpu.grid.sizes()})},
			.shared_memory_bytes{0}
			};

		utils::cuda::device::call(&::step, kernel_params, data_gpu.grid_kernel_side, data_cpu.time);

		utils::cuda::device::call(&::step_distribution, kernel_params, data_gpu.grid_kernel_side, utils::math::vec2u{-1, -1});
		utils::cuda::device::call(&::step_distribution, kernel_params, data_gpu.grid_kernel_side, utils::math::vec2u{ 0, -1});
		utils::cuda::device::call(&::step_distribution, kernel_params, data_gpu.grid_kernel_side, utils::math::vec2u{ 1, -1});
		utils::cuda::device::call(&::step_distribution, kernel_params, data_gpu.grid_kernel_side, utils::math::vec2u{-1,  0});
		utils::cuda::device::call(&::step_distribution, kernel_params, data_gpu.grid_kernel_side, utils::math::vec2u{ 1,  0});
		utils::cuda::device::call(&::step_distribution, kernel_params, data_gpu.grid_kernel_side, utils::math::vec2u{-1,  1});
		utils::cuda::device::call(&::step_distribution, kernel_params, data_gpu.grid_kernel_side, utils::math::vec2u{ 0,  1});
		utils::cuda::device::call(&::step_distribution, kernel_params, data_gpu.grid_kernel_side, utils::math::vec2u{ 1,  1});

		utils::cuda::device::call(&::step_recover_undistributed, kernel_params, data_gpu.grid_kernel_side);
		}

	void game::build(float absorption) noexcept
		{
		//data_cpu.occupied_mask[data_cpu.mouse_tile] = true;
		utils::cuda::device::call(&::build_on_tile, {.threads{.per_block{1u}, .blocks{1u}}}, data_gpu.grid_kernel_side, data_cpu.mouse_tile, absorption);
		}

	void game::cpu_to_gpu() noexcept
		{
		data_gpu = ::game::data_gpu{data_cpu};
		}
	void game::gpu_to_cpu() noexcept
		{
		thrust::copy(data_gpu.grid.begin(), data_gpu.grid.end(), data_cpu.grid.begin());
		}
	}