#include "hip/hip_runtime.h"
﻿#include <cmath>
#include <cstddef>
#include <algorithm>

#include <utils/math/vec2.h>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <SFML/Graphics.hpp>
#include "glew.h"
#include <SFML/OpenGL.hpp>

#include <GL/GL.h>
#include "cuda_gl_interop.h"






struct cuda_gl_interop_texture_stuff
	{
	utils::math::vec2s texture_size;

	unsigned int size_tex_data;
	unsigned int num_texels;
	unsigned int num_values;

	GLuint tex_cudaResult;

	GLuint pbo_dest;
	hipGraphicsResource* cuda_pbo_dest_resource;

	cuda_gl_interop_texture_stuff(utils::math::vec2s texture_size) : texture_size{texture_size} { initGLBuffers(); }

	void initGLBuffers() 
		{
		// create pbo
		createPBO(&pbo_dest, &cuda_pbo_dest_resource);
		}

	void createPBO(GLuint* pbo, struct hipGraphicsResource** pbo_resource) {
	  // set up vertex data parameter
		num_texels = texture_size.x * texture_size.y;
		num_values = num_texels * 4;
		size_tex_data = sizeof(GLubyte) * num_values;
		void* data = malloc(size_tex_data);

		// create buffer object
		glGenBuffers(1, pbo);
		glBindBuffer(GL_ARRAY_BUFFER, *pbo);
		glBufferData(GL_ARRAY_BUFFER, size_tex_data, data, GL_DYNAMIC_DRAW);
		free(data);

		glBindBuffer(GL_ARRAY_BUFFER, 0);
		
		// register this buffer object with CUDA
		hipGraphicsGLRegisterBuffer(pbo_resource, *pbo, cudaGraphicsMapFlagsNone);
		}

	void deletePBO(GLuint* pbo) {
		glDeleteBuffers(1, pbo);
		*pbo = 0;
		}
	};


__global__ void kernel(std::byte* g_odata, utils::math::vec2s texture_size)
	{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bw = blockDim.x;
	int bh = blockDim.y;
	int x = blockIdx.x * bw + tx;
	int y = blockIdx.y * bh + ty;

	if (x > texture_size.x || y > texture_size.y) { return; }

	size_t base_index{(y * texture_size.x + x) * 4};

	float r{x / static_cast<float>(texture_size.x)};
	float g{y / static_cast<float>(texture_size.y)};
	float b{1.f};
	float a{1.f};

	g_odata[base_index + 0] = static_cast<std::byte>(r * 255.f);
	g_odata[base_index + 1] = static_cast<std::byte>(g * 255.f);
	g_odata[base_index + 2] = static_cast<std::byte>(b * 255.f);
	g_odata[base_index + 3] = static_cast<std::byte>(a * 255.f);
	}



#include <iostream>
int main()
	{
	sf::Context context;
	context.setActive(true);
	glewInit(); //glewInit MUST be called after initializing a context, wether real or unused. Otherwise opengl functions won't be available

	utils::math::vec2s texture_size{805, 600};

	sf::Texture texture;
	texture.create(texture_size.x, texture_size.y);

	cuda_gl_interop_texture_stuff cgits{texture_size};

	// calculate grid size
	dim3 block(16, 16, 1);

	dim3 grid
		{
		static_cast<unsigned int>(std::ceil(static_cast<float>(texture_size.x) / static_cast<float>(block.x))),
		static_cast<unsigned int>(std::ceil(static_cast<float>(texture_size.y) / static_cast<float>(block.y))),
		static_cast<unsigned int>(1)
		};

	if (true)
		{
		std::byte* out_data;
		hipGraphicsMapResources(1, &cgits.cuda_pbo_dest_resource, 0);
		size_t num_bytes;
		hipGraphicsResourceGetMappedPointer((void**)&out_data, &num_bytes, cgits.cuda_pbo_dest_resource);

		kernel<<<grid, block>>>(out_data, texture_size);

		hipDeviceSynchronize();

		hipGraphicsUnmapResources(1, &cgits.cuda_pbo_dest_resource, 0);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, cgits.pbo_dest);

		glBindTexture(GL_TEXTURE_2D, texture.getNativeHandle());
		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, texture_size.x, texture_size.y, GL_RGBA,
			GL_UNSIGNED_BYTE, NULL);

		glBindBuffer(GL_PIXEL_PACK_BUFFER_ARB, 0);
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
		}

	auto image{texture.copyToImage()};
	image.saveToFile("hello.png");

	return 0;
	}
